#include "hip/hip_runtime.h"
#include "utility.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "numgen.c"


__host__ //Runs on the CPU (host)
void errorexit(const char *s) {
    printf("\n%s", s);
    exit(EXIT_FAILURE);
}


__device__ //Runs on the GPU (device)
int isPrime(unsigned long int n) {
    if (n <= 1) return 0;
    if (n <= 3) return 1;
    if (n % 2 == 0 || n % 3 == 0) return 0;
    for (unsigned long int i = 5; i * i <= n; i += 6) {
        if (n % i == 0 || n % (i + 2) == 0)
            return 0;
    }
    return 1;
}


__global__ //Runs on the GPU (device) but is launched from the CPU
void countPrimes(unsigned long int *numbers, int *count, long size) {
    long idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        if (isPrime(numbers[idx])) {
            atomicAdd(count, 1);
        }
    }
}


int main(int argc,char **argv) {

  Args ins__args;
  parseArgs(&ins__args, &argc, argv);
  
  //program input argument
  long inputArgument = ins__args.arg; 
  unsigned long int *numbers = (unsigned long int*)malloc(inputArgument * sizeof(unsigned long int));
  numgen(inputArgument, numbers);

  struct timeval ins__tstart, ins__tstop;
  gettimeofday(&ins__tstart, NULL);
  
  // run your CUDA kernel(s) here
  
  unsigned long int *device_numbers = NULL;
  int *device_prime_count = NULL;
  int host_prime_count = 0;
  int threadsinblock=1024;
  int blocksingrid= (threadsinblock + inputArgument -1)/threadsinblock;	
  
  if (hipSuccess != hipMalloc((void **)&device_numbers, inputArgument * sizeof(unsigned long int))) {
  	errorexit("Error allocating device memory for numbers array");
  }
  if (hipSuccess != hipMalloc((void **)&device_prime_count, sizeof(int))) {
  	errorexit("Error allocating device memory for prime counter");
  }
  if (hipSuccess != hipMemcpy(device_numbers, numbers, inputArgument * sizeof(unsigned long int), hipMemcpyHostToDevice)) {
  	errorexit("Error copying numbers to device");
  }
    
	countPrimes<<<blocksingrid, threadsinblock>>>(device_numbers, device_prime_count, inputArgument);

  // synchronize/finalize your CUDA computations
  
  hipDeviceSynchronize();
  hipMemcpy(&host_prime_count, device_prime_count, sizeof(int), hipMemcpyDeviceToHost);
	
	printf("\nPrime numbers: %d\n", host_prime_count);

  hipFree(device_numbers);
	free(numbers);

  gettimeofday(&ins__tstop, NULL);
  ins__printtime(&ins__tstart, &ins__tstop, ins__args.marker);

  return 0;
}
